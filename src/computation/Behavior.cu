﻿#include "computation/Behavior.h"
#include "computation/Kernels.h"
#include <stdio.h>
#include <chrono>
#include <thread>
#include <cuda_gl_interop.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "Config.hpp"

namespace computation {

    Behavior::Behavior(GLuint shoalBuffer, FishProperties properties) : _shoalBuffer(shoalBuffer) { //TODO FREE
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return;
        }

        cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
            return;
        }

        // Allocate GPU buffers fo fish properties
        cudaStatus = hipMalloc(&_propertiesDevice, sizeof(FishProperties));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
        cudaStatus = hipMemcpy(_propertiesDevice, &properties, sizeof(FishProperties), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish velocities
        cudaStatus = hipMalloc(&_velocitiesDevice, sizeof(FishShoalVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
        FishShoalVelocities velocities;
        std::fill(velocities.velocityX, velocities.velocityX + Config::SHOAL_SIZE, 0.001f);
        std::fill(velocities.velocityY, velocities.velocityY + Config::SHOAL_SIZE, 0.000f);
        std::fill(velocities.velocityZ, velocities.velocityZ + Config::SHOAL_SIZE, 0.000f);
        cudaStatus = hipMemcpy(_velocitiesDevice, &velocities, sizeof(FishShoalVelocities), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish ids
        cudaStatus = hipMalloc(&_fishIdsDevice, Config::SHOAL_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        // Allocate GPU buffers for region indexes
		cudaStatus = hipMalloc(&_regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int) * 2);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }

        // Allocate GPU buffers for region starts
        cudaStatus = hipMalloc(&_regionStartsDevice, Config::REGION_COUNT * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        cudaStatus = ComputeRegionsCheatSheet();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
    }

    Behavior::~Behavior() {}

    hipError_t Behavior::ComputeRegionsCheatSheet()
    {
        hipError_t cudaStatus;
        // Allocate GPU buffers for regions cheat sheet
        cudaStatus = hipMalloc(&_regionsCheatSheetDevice, Config::REGION_COUNT * 27 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        int regionsCheatSheet[Config::REGION_COUNT * 27];
        int dim = Config::REGION_DIM_COUNT;
        for (int x = 0; x < dim; x++)
        {
            for (int y = 0; y < dim; y++)
            {
                for (int z = 0; z < dim; z++)
                {
                    int globalIndex = x * dim * dim + y * dim + z;
                    int regionsToCheckIndex = 0;

                    for (int i = -1; i <= 1; i++)
                    {
                        for (int j = -1; j <= 1; j++)
                        {
                            for (int k = -1; k <= 1; k++)
                            {
                                int xIndex = x + i;
                                int yIndex = y + j;
                                int zIndex = z + k;

                                if (xIndex >= 0 && xIndex < dim && yIndex >= 0 && yIndex < dim && zIndex >= 0 && zIndex < dim)
                                {
                                    regionsCheatSheet[globalIndex * 27 + regionsToCheckIndex] = xIndex * dim * dim + yIndex * dim + zIndex;
                                    regionsToCheckIndex++;
                                }
                            }
                        }
                    }

                    for (int i = regionsToCheckIndex; i < 27; i++)
					{
						regionsCheatSheet[globalIndex * 27 + i] = -1;
					}
                }
            }
        }
        
        cudaStatus = hipMemcpy(_regionsCheatSheetDevice, regionsCheatSheet, Config::REGION_COUNT * 27 * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return cudaStatus;
		}

        return hipSuccess;
    }

    hipError_t Behavior::ComputeMove()
    {
        hipError_t cudaStatus;

        cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsMapResources failed!");
            return cudaStatus;
        }

        void* positions_dev;
        size_t size;
        cudaStatus = hipGraphicsResourceGetMappedPointer(&positions_dev, &size, _resource);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
            return cudaStatus;
        }

        // Assign fish to regions
        assignFishToRegionsKernel << <1, Config::SHOAL_SIZE >> > ((float*)positions_dev, _fishIdsDevice, _regionIndexesDevice);
        thrust::sort_by_key(thrust::device, _regionIndexesDevice, _regionIndexesDevice + Config::SHOAL_SIZE, _fishIdsDevice);
        findRegionStartsKernel << <1, Config::SHOAL_SIZE >> > (_fishIdsDevice, _regionIndexesDevice, _regionStartsDevice);

        // Compute movement
        computeMoveKernel << <1, Config::SHOAL_SIZE >> > ((float*)positions_dev, _velocitiesDevice, _propertiesDevice, _fishIdsDevice, _regionIndexesDevice, _regionStartsDevice, _regionsCheatSheetDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        float output4[Config::SHOAL_SIZE * 3];
        cudaStatus = hipMemcpy(output4, positions_dev, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        // for debugging
        int output[Config::SHOAL_SIZE * 2];
        cudaStatus = hipMemcpy(output, _regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int) * 2, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        // for debugging
        int output2[Config::SHOAL_SIZE];
        cudaStatus = hipMemcpy(output2, _fishIdsDevice, Config::SHOAL_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        // for debugging
        int output3[Config::REGION_COUNT];
        cudaStatus = hipMemcpy(output3, _regionStartsDevice, Config::REGION_COUNT * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        // for debugging
        int output5[Config::REGION_COUNT * 27];
        cudaStatus = hipMemcpy(output5, _regionsCheatSheetDevice, Config::REGION_COUNT * 27 * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        FishShoalVelocities velocities;
        cudaStatus = hipMemcpy(&velocities, _velocitiesDevice, sizeof(velocities), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsUnmapResources failed!");
            return cudaStatus;
        }

        return cudaStatus;
    }
}
