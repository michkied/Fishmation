﻿#include "computation/Behavior.h"
#include "computation/Kernels.h"
#include "Config.hpp"

#include <chrono>
#include <cuda_gl_interop.h>
#include <random>
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

namespace computation
{

	Behavior::Behavior(Config& config, GLuint shoalBuffer, FishProperties& properties) : _shoalBuffer(shoalBuffer), _propertiesHost(properties), _config(config)
	{
		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			return;
		}

		cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
			return;
		}

		// Allocate GPU buffers to general config
		cudaStatus = hipMalloc(&_configDevice, sizeof(Config));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
		cudaStatus = hipMemcpy(_configDevice, &config, sizeof(Config), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}

		// Allocate GPU buffers fo fish properties
		cudaStatus = hipMalloc(&_propertiesDevice, sizeof(FishProperties));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
		cudaStatus = hipMemcpy(_propertiesDevice, &properties, sizeof(FishProperties), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}

		// Allocate GPU buffers for fish velocities
		cudaStatus = hipMalloc(&_velocitiesDevice, _config.FISH_COUNT * 3 * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
		cudaStatus = hipMemset(_velocitiesDevice, 0, _config.FISH_COUNT * 3 * sizeof(float)); // regular fish start with speed 0

		// Generate speeds for predators
		float* predatorsVx = new float[_config.PREDATOR_COUNT];
		float* predatorsVy = new float[_config.PREDATOR_COUNT];
		float* predatorsVz = new float[_config.PREDATOR_COUNT];
		std::random_device rd;
		std::mt19937 gen(rd());
		std::uniform_real_distribution<float> distribution(_config.PREDATOR_MIN_SPEED, _config.PREDATOR_MAX_SPEED);
		for (int i = 0; i < _config.PREDATOR_COUNT; i++) {
			predatorsVx[i] = distribution(gen);
			predatorsVy[i] = distribution(gen);
			predatorsVz[i] = distribution(gen);
		}
		// Overwrite 0s for predators
		cudaStatus = hipMemcpy(&_velocitiesDevice[_config.SHOAL_SIZE], predatorsVx, sizeof(float) * _config.PREDATOR_COUNT, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}
		cudaStatus = hipMemcpy(&_velocitiesDevice[_config.FISH_COUNT + _config.SHOAL_SIZE], predatorsVy, sizeof(float) * _config.PREDATOR_COUNT, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}
		cudaStatus = hipMemcpy(&_velocitiesDevice[_config.FISH_COUNT * 2 + _config.SHOAL_SIZE], predatorsVz, sizeof(float) * _config.PREDATOR_COUNT, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}
		delete[] predatorsVx;
		delete[] predatorsVy;
		delete[] predatorsVz;

		// Allocate GPU buffers for fish ids
		cudaStatus = hipMalloc(&_fishIdsDevice, _config.SHOAL_SIZE * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

		// Allocate GPU buffers for region indexes
		cudaStatus = hipMalloc(&_regionIndexesDevice, _config.SHOAL_SIZE * sizeof(int) * 2);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

		// Allocate GPU buffers for region starts
		cudaStatus = hipMalloc(&_regionStartsDevice, _config.REGION_COUNT * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

		// Precompute lookup table for neighboring regions
		cudaStatus = ComputeRegionsCheatSheet();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "ComputeRegionsCheatSheet failed!");
			return;
		}
	}

	Behavior::~Behavior()
	{
		hipFree(_configDevice);
		hipFree(_propertiesDevice);
		hipFree(_velocitiesDevice);
		hipFree(_fishIdsDevice);
		hipFree(_regionIndexesDevice);
		hipFree(_regionStartsDevice);
		hipFree(_regionsCheatSheetDevice);

		hipError_t cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
		}
	}

	hipError_t Behavior::ComputeRegionsCheatSheet()
	{
		hipError_t cudaStatus;
		// Allocate GPU buffers for regions cheat sheet
		cudaStatus = hipMalloc(&_regionsCheatSheetDevice, _config.REGION_COUNT * 27 * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return cudaStatus;
		}

		computeRegionsCheatSheetKernel << < _config.REGION_COUNT / _config.THREADS_PER_BLOCK + 1, _config.THREADS_PER_BLOCK >> > (_configDevice, _regionsCheatSheetDevice);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "computeRegionsCheatSheetKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeRegionsCheatSheetKernel!\n", cudaStatus);
			return cudaStatus;
		}

		return hipSuccess;
	}

	hipError_t Behavior::ComputeMove()
	{
		hipError_t cudaStatus;

		if (_propertiesHost.changeCounter != _propertiesChangeCounter) {
			cudaStatus = hipMemcpy(_propertiesDevice, &_propertiesHost, sizeof(FishProperties), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				return cudaStatus;
			}
			_propertiesChangeCounter = _propertiesHost.changeCounter;
		}

		cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsMapResources failed!");
			return cudaStatus;
		}

		void* positions_dev;
		size_t size;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&positions_dev, &size, _resource);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
			return cudaStatus;
		}

		// Assign fish to regions
		assignFishToRegionsKernel << < _config.SHOAL_SIZE / _config.THREADS_PER_BLOCK + 1, _config.THREADS_PER_BLOCK >> > (_configDevice, (float*)positions_dev, _fishIdsDevice, _regionIndexesDevice);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "assignFishToRegionsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching assignFishToRegionsKernel!\n", cudaStatus);
			return cudaStatus;
		}
		thrust::sort_by_key(thrust::device, _regionIndexesDevice, _regionIndexesDevice + _config.SHOAL_SIZE, _fishIdsDevice);
		findRegionStartsKernel << < _config.SHOAL_SIZE / _config.THREADS_PER_BLOCK + 1, _config.THREADS_PER_BLOCK >> > (_configDevice, _fishIdsDevice, _regionIndexesDevice, _regionStartsDevice);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "findRegionStartsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findRegionStartsKernel!\n", cudaStatus);
			return cudaStatus;
		}

		// Compute shoal movement
		computeShoalMoveKernel << < _config.SHOAL_SIZE / _config.THREADS_PER_BLOCK + 1, _config.THREADS_PER_BLOCK >> > (_configDevice, (float*)positions_dev, _velocitiesDevice, _propertiesDevice, _fishIdsDevice, _regionIndexesDevice, _regionStartsDevice, _regionsCheatSheetDevice);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "computeShoalMoveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeShoalMoveKernel!\n", cudaStatus);
			return cudaStatus;
		}

		// Compute predator movement
		computePredatorMoveKernel << < _config.PREDATOR_COUNT / _config.THREADS_PER_BLOCK + 1, _config.THREADS_PER_BLOCK >> > (_configDevice, (float*)positions_dev, _velocitiesDevice);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "computePredatorMoveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computePredatorMoveKernel!\n", cudaStatus);
			return cudaStatus;
		}

		cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsUnmapResources failed!");
			return cudaStatus;
		}

		return cudaStatus;
	}
}
