﻿#include "computation/Behavior.h"
#include "computation/Kernel.h"
#include <stdio.h>
#include <chrono>
#include <thread>
#include <cuda_gl_interop.h>
#include "Config.hpp"

namespace computation {

    Behavior::Behavior(GLuint shoalBuffer) : _shoalBuffer(shoalBuffer) {
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return;
        }

        cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
            return;
        }
    }

    Behavior::~Behavior() {}

    hipError_t Behavior::ComputeMove()
    {
        hipError_t cudaStatus;

        cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsMapResources failed!");
            return cudaStatus;
        }

        void* dev_ptr;
        size_t size;
        cudaStatus = hipGraphicsResourceGetMappedPointer(&dev_ptr, &size, _resource);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
            return cudaStatus;
        }

        // Launch a kernel on the GPU with one thread for each element.
        computeMoveKernel << <1, Config::SHOAL_SIZE >> > ((float*)dev_ptr);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        float output[Config::SHOAL_SIZE * 3];
        cudaStatus = hipMemcpy(output, dev_ptr, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsUnmapResources failed!");
            return cudaStatus;
        }

        return cudaStatus;
    }
}
