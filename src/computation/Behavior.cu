﻿#include "Config.hpp"
#include "computation/Behavior.h"
#include "computation/Kernels.h"

#include <stdio.h>
#include <chrono>
#include <thread>
#include <cuda_gl_interop.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

namespace computation {

    Behavior::Behavior(GLuint shoalBuffer, FishProperties properties) : _shoalBuffer(shoalBuffer) { //TODO FREE
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return;
        }

        cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
            return;
        }

        // Allocate GPU buffers fo fish properties
        cudaStatus = hipMalloc(&_propertiesDevice, sizeof(FishProperties));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
        cudaStatus = hipMemcpy(_propertiesDevice, &properties, sizeof(FishProperties), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish velocities
        cudaStatus = hipMalloc(&_velocitiesDevice, sizeof(FishShoalVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
        FishShoalVelocities velocities;
        std::fill(velocities.velocityX, velocities.velocityX + Config::SHOAL_SIZE, 0.0f);
        std::fill(velocities.velocityY, velocities.velocityY + Config::SHOAL_SIZE, 0.0f);
        std::fill(velocities.velocityZ, velocities.velocityZ + Config::SHOAL_SIZE, 0.0f);
        cudaStatus = hipMemcpy(_velocitiesDevice, &velocities, sizeof(FishShoalVelocities), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish ids
        cudaStatus = hipMalloc(&_fishIdsDevice, Config::SHOAL_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        // Allocate GPU buffers for region indexes
		cudaStatus = hipMalloc(&_regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int) * 2);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }

        // Allocate GPU buffers for region starts
        cudaStatus = hipMalloc(&_regionStartsDevice, Config::REGION_COUNT * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        cudaStatus = ComputeRegionsCheatSheet();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "ComputeRegionsCheatSheet failed!");
            return;
        }

        cudaStatus = SetupPredators();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "SetupPredators failed!");
            return;
        }
    }

    Behavior::~Behavior() {
        hipFree(_propertiesDevice);
		hipFree(_velocitiesDevice);
		hipFree(_fishIdsDevice);
		hipFree(_regionIndexesDevice);
		hipFree(_regionStartsDevice);
		hipFree(_regionsCheatSheetDevice);
        hipFree(_predatorStateDevice);
        hipFree(_predatorVelocitiesDevice);
    }

    hipError_t Behavior::ComputeRegionsCheatSheet()
    {
        hipError_t cudaStatus;
        // Allocate GPU buffers for regions cheat sheet
        cudaStatus = hipMalloc(&_regionsCheatSheetDevice, Config::REGION_COUNT * 27 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        computeRegionsCheatSheetKernel<<< Config::REGION_COUNT / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>(_regionsCheatSheetDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        return hipSuccess;
    }

    hipError_t Behavior::SetupPredators() {
        hipError_t cudaStatus;
        cudaStatus = hipMalloc(&_predatorStateDevice, Config::PREDATOR_COUNT * sizeof(hiprandState));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc(&_predatorVelocitiesDevice, sizeof(PredatorVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        setupPredatorRandomnessKernel << < Config::PREDATOR_COUNT / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >> > (_predatorStateDevice, _predatorVelocitiesDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        return hipSuccess;
    }

    hipError_t Behavior::ComputeMove()
    {
        hipError_t cudaStatus;

        cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsMapResources failed!");
            return cudaStatus;
        }

        void* positions_dev;
        size_t size;
        cudaStatus = hipGraphicsResourceGetMappedPointer(&positions_dev, &size, _resource);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
            return cudaStatus;
        }

        // Assign fish to regions
        assignFishToRegionsKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>((float*)positions_dev, _fishIdsDevice, _regionIndexesDevice);
        thrust::sort_by_key(thrust::device, _regionIndexesDevice, _regionIndexesDevice + Config::SHOAL_SIZE, _fishIdsDevice);
        findRegionStartsKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>(_fishIdsDevice, _regionIndexesDevice, _regionStartsDevice);

        // Compute shoal movement
        computeShoalMoveKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>((float*)positions_dev, _velocitiesDevice, _propertiesDevice, _fishIdsDevice, _regionIndexesDevice, _regionStartsDevice, _regionsCheatSheetDevice);
        computePredatorMoveKernel << < Config::PREDATOR_COUNT / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >> >((float*)positions_dev, _predatorVelocitiesDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        //float output4[Config::SHOAL_SIZE * 3];
        //cudaStatus = hipMemcpy(output4, positions_dev, size, hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}

        //// for debugging
        //int output[Config::SHOAL_SIZE * 2];
        //cudaStatus = hipMemcpy(output, _regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int) * 2, hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}
        //// for debugging
        //int output2[Config::SHOAL_SIZE];
        //cudaStatus = hipMemcpy(output2, _fishIdsDevice, Config::SHOAL_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}
        //// for debugging
        //int output3[Config::REGION_COUNT];
        //cudaStatus = hipMemcpy(output3, _regionStartsDevice, Config::REGION_COUNT * sizeof(int), hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}
        //// for debugging
        //int output5[Config::REGION_COUNT * 27];
        //cudaStatus = hipMemcpy(output5, _regionsCheatSheetDevice, Config::REGION_COUNT * 27 * sizeof(int), hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}

        //FishShoalVelocities velocities;
        //cudaStatus = hipMemcpy(&velocities, _velocitiesDevice, sizeof(velocities), hipMemcpyDeviceToHost);
        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "hipMemcpy failed!");
        //    return cudaStatus;
        //}

        cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsUnmapResources failed!");
            return cudaStatus;
        }

        return cudaStatus;
    }
}
