﻿#include "Config.hpp"
#include "computation/Behavior.h"
#include "computation/Kernels.h"

#include <stdio.h>
#include <chrono>
#include <random>
#include <cuda_gl_interop.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

namespace computation {

    Behavior::Behavior(GLuint shoalBuffer, FishProperties& properties) : _shoalBuffer(shoalBuffer), _propertiesHost(properties) {
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return;
        }

        cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
            return;
        }

        // Allocate GPU buffers fo fish properties
        cudaStatus = hipMalloc(&_propertiesDevice, sizeof(FishProperties));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
        cudaStatus = hipMemcpy(_propertiesDevice, &properties, sizeof(FishProperties), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish velocities
        cudaStatus = hipMalloc(&_velocitiesDevice, sizeof(FishShoalVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
        FishShoalVelocities velocities;
        std::fill(velocities.velocityX, velocities.velocityX + Config::SHOAL_SIZE, 0.0f);
        std::fill(velocities.velocityY, velocities.velocityY + Config::SHOAL_SIZE, 0.0f);
        std::fill(velocities.velocityZ, velocities.velocityZ + Config::SHOAL_SIZE, 0.0f);
        cudaStatus = hipMemcpy(_velocitiesDevice, &velocities, sizeof(FishShoalVelocities), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish ids
        cudaStatus = hipMalloc(&_fishIdsDevice, Config::SHOAL_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        // Allocate GPU buffers for region indexes
		cudaStatus = hipMalloc(&_regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int) * 2);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }

        // Allocate GPU buffers for region starts
        cudaStatus = hipMalloc(&_regionStartsDevice, Config::REGION_COUNT * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        // Precompute lookup table for neighboring regions
        cudaStatus = ComputeRegionsCheatSheet();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "ComputeRegionsCheatSheet failed!");
            return;
        }

        // Allocate GPU buffers for predator velocities
        cudaStatus = hipMalloc(&_predatorVelocitiesDevice, sizeof(PredatorVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> distribution(Config::PREDATOR_MIN_SPEED, Config::PREDATOR_MAX_SPEED);
        PredatorVelocities predatorVelocities;
        for (int i = 0; i < Config::PREDATOR_COUNT; i++) {
			predatorVelocities.velocityX[i] = distribution(gen);
			predatorVelocities.velocityY[i] = distribution(gen);
			predatorVelocities.velocityZ[i] = distribution(gen);
		}
        cudaStatus = hipMemcpy(_predatorVelocitiesDevice, &predatorVelocities, sizeof(PredatorVelocities), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}
    }

    Behavior::~Behavior() {
        hipFree(_propertiesDevice);
		hipFree(_velocitiesDevice);
		hipFree(_fishIdsDevice);
		hipFree(_regionIndexesDevice);
		hipFree(_regionStartsDevice);
		hipFree(_regionsCheatSheetDevice);
        hipFree(_predatorVelocitiesDevice);

        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
        }
    }

    hipError_t Behavior::ComputeRegionsCheatSheet()
    {
        hipError_t cudaStatus;
        // Allocate GPU buffers for regions cheat sheet
        cudaStatus = hipMalloc(&_regionsCheatSheetDevice, Config::REGION_COUNT * 27 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        computeRegionsCheatSheetKernel<<< Config::REGION_COUNT / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>(_regionsCheatSheetDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "computeRegionsCheatSheetKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeRegionsCheatSheetKernel!\n", cudaStatus);
            return cudaStatus;
        }

        return hipSuccess;
    }

    hipError_t Behavior::ComputeMove()
    {
        hipError_t cudaStatus;

        if (_propertiesHost.changeCounter != _propertiesChangeCounter) {
            cudaStatus = hipMemcpy(_propertiesDevice, &_propertiesHost, sizeof(FishProperties), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                return cudaStatus;
            }
            _propertiesChangeCounter = _propertiesHost.changeCounter;
        }

        cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsMapResources failed!");
            return cudaStatus;
        }

        void* positions_dev;
        size_t size;
        cudaStatus = hipGraphicsResourceGetMappedPointer(&positions_dev, &size, _resource);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
            return cudaStatus;
        }

        // Assign fish to regions
        assignFishToRegionsKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>((float*)positions_dev, _fishIdsDevice, _regionIndexesDevice);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "assignFishToRegionsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching assignFishToRegionsKernel!\n", cudaStatus);
            return cudaStatus;
        }
        thrust::sort_by_key(thrust::device, _regionIndexesDevice, _regionIndexesDevice + Config::SHOAL_SIZE, _fishIdsDevice);
        findRegionStartsKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>(_fishIdsDevice, _regionIndexesDevice, _regionStartsDevice);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "findRegionStartsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findRegionStartsKernel!\n", cudaStatus);
            return cudaStatus;
        }

        // Compute shoal movement
        computeShoalMoveKernel<<< Config::SHOAL_SIZE / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >>>((float*)positions_dev, _velocitiesDevice, _propertiesDevice, _fishIdsDevice, _regionIndexesDevice, _regionStartsDevice, _regionsCheatSheetDevice);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "computeShoalMoveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeShoalMoveKernel!\n", cudaStatus);
            return cudaStatus;
        }

        // Compute predator movement
        computePredatorMoveKernel << < Config::PREDATOR_COUNT / Config::THREADS_PER_BLOCK + 1, Config::THREADS_PER_BLOCK >> >((float*)positions_dev, _predatorVelocitiesDevice);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "computePredatorMoveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computePredatorMoveKernel!\n", cudaStatus);
            return cudaStatus;
        }

        cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsUnmapResources failed!");
            return cudaStatus;
        }

        return cudaStatus;
    }
}
