﻿#include "computation/Behavior.h"
#include "computation/Kernels.h"
#include <stdio.h>
#include <chrono>
#include <thread>
#include <cuda_gl_interop.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "Config.hpp"

namespace computation {

    Behavior::Behavior(GLuint shoalBuffer, FishProperties properties) : _shoalBuffer(shoalBuffer) { //TODO FREE
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return;
        }

        cudaStatus = hipGraphicsGLRegisterBuffer(&_resource, _shoalBuffer, hipGraphicsRegisterFlagsNone);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsGLRegisterBuffer failed!");
            return;
        }

        // Allocate GPU buffers fo fish properties
        cudaStatus = hipMalloc(&_propertiesDevice, sizeof(FishProperties));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
        cudaStatus = hipMemcpy(_propertiesDevice, &properties, sizeof(FishProperties), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish velocities
        cudaStatus = hipMalloc(&_velocitiesDevice, sizeof(FishShoalVelocities));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }
        FishShoalVelocities velocities;
        std::fill(velocities.velocityX, velocities.velocityX + Config::SHOAL_SIZE, 0.005f);
        std::fill(velocities.velocityY, velocities.velocityY + Config::SHOAL_SIZE, 0.005f);
        std::fill(velocities.velocityZ, velocities.velocityZ + Config::SHOAL_SIZE, 0.005f);
        cudaStatus = hipMemcpy(_velocitiesDevice, &velocities, sizeof(FishShoalVelocities), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }

        // Allocate GPU buffers for fish ids
        cudaStatus = hipMalloc(&_fishIdsDevice, Config::SHOAL_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}

        // Allocate GPU buffers for region indexes
		cudaStatus = hipMalloc(&_regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return;
        }

        // Allocate GPU buffers for region starts
        cudaStatus = hipMalloc(&_regionStartsDevice, Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT * sizeof(int));
        if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return;
		}
    }

    Behavior::~Behavior() {}

    hipError_t Behavior::ComputeMove()
    {
        hipError_t cudaStatus;

        cudaStatus = hipGraphicsMapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsMapResources failed!");
            return cudaStatus;
        }

        void* positions_dev;
        size_t size;
        cudaStatus = hipGraphicsResourceGetMappedPointer(&positions_dev, &size, _resource);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer failed!");
            return cudaStatus;
        }

        // Assign fish to regions
        assignFishToRegionsKernel << <1, Config::SHOAL_SIZE >> > ((float*)positions_dev, _fishIdsDevice, _regionIndexesDevice);
        thrust::sort_by_key(thrust::device, _regionIndexesDevice, _regionIndexesDevice + Config::SHOAL_SIZE, _fishIdsDevice);
        findRegionStartsKernel << <1, Config::SHOAL_SIZE >> > (_fishIdsDevice, _regionIndexesDevice, _regionStartsDevice);

        computeMoveKernel << <1, Config::SHOAL_SIZE >> > ((float*)positions_dev, _velocitiesDevice, _propertiesDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        float output4[Config::SHOAL_SIZE * 3];
        cudaStatus = hipMemcpy(output4, positions_dev, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        // for debugging
        int output[Config::SHOAL_SIZE];
        cudaStatus = hipMemcpy(output, _regionIndexesDevice, Config::SHOAL_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        // for debugging
        int output2[Config::SHOAL_SIZE];
        cudaStatus = hipMemcpy(output2, _fishIdsDevice, Config::SHOAL_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        // for debugging
        int output3[Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT];
        cudaStatus = hipMemcpy(output3, _regionStartsDevice, Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipGraphicsUnmapResources(1, &_resource, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsUnmapResources failed!");
            return cudaStatus;
        }

        return cudaStatus;
    }
}
