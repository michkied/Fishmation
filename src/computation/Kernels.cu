#include "hip/hip_runtime.h"
#include "computation/Kernels.h"
#include <limits>
#include <cmath>

namespace computation {
    __global__ void computeRegionsCheatSheetKernel(int* regionsCheatSheet)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= Config::REGION_COUNT) return;

        int dim = Config::REGION_DIM_COUNT;
		int x = i % dim;
		int y = (i / dim) % dim;
		int z = i / (dim * dim);

        int globalIndex = x * dim * dim + y * dim + z;
        int regionsToCheckIndex = 0;

        for (int i = -1; i <= 1; i++)
        {
            for (int j = -1; j <= 1; j++)
            {
                for (int k = -1; k <= 1; k++)
                {
                    int xIndex = x + i;
                    int yIndex = y + j;
                    int zIndex = z + k;

                    if (xIndex >= 0 && xIndex < dim && yIndex >= 0 && yIndex < dim && zIndex >= 0 && zIndex < dim)
                    {
                        regionsCheatSheet[globalIndex * 27 + regionsToCheckIndex] = xIndex * dim * dim + yIndex * dim + zIndex;
                        regionsToCheckIndex++;
                    }
                }
            }
        }

        for (int i = regionsToCheckIndex; i < 27; i++)
        {
            regionsCheatSheet[globalIndex * 27 + i] = -1;
        }
	}

    __global__ void assignFishToRegionsKernel(float* positions, int* fishIds, int* regionIndexes)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= Config::SHOAL_SIZE) return;
        
        int xIndex = i;
        int yIndex = i + Config::FISH_COUNT;
        int zIndex = i + Config::FISH_COUNT * 2;

        float tempX = positions[xIndex] / Config::REGION_SIZE;
        float tempY = positions[yIndex] / Config::REGION_SIZE;
        float tempZ = positions[zIndex] / Config::REGION_SIZE;

        int idX = (tempX >= 0) ? (int)tempX + 1 : (int)tempX - 1;
        int idY = (tempY >= 0) ? (int)tempY + 1 : (int)tempY - 1;
        int idZ = (tempZ >= 0) ? (int)tempZ + 1 : (int)tempZ - 1;

        int linearX = Config::REGION_DIM_COUNT / 2 - idX - (int)(idX < 0);
        int linearY = Config::REGION_DIM_COUNT / 2 - idY - (int)(idY < 0);
        int linearZ = Config::REGION_DIM_COUNT / 2 - idZ - (int)(idZ < 0);

        int index = linearX + linearY * Config::REGION_DIM_COUNT + linearZ * Config::REGION_DIM_COUNT * Config::REGION_DIM_COUNT;
        regionIndexes[i] = index;
        regionIndexes[i + Config::SHOAL_SIZE] = index;  // this copy won't be sorted (for checking fish region in computeMoveKernel)
        fishIds[i] = i;
    }

    __global__ void findRegionStartsKernel(int* fishIds, int* regionIndexes, int* regionStarts) 
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= Config::SHOAL_SIZE) return;

		if (i == 0) {
			regionStarts[regionIndexes[i]] = 0;
		}
		else if (regionIndexes[i] != regionIndexes[i - 1]) {
			regionStarts[regionIndexes[i]] = i;
		}
    }

    __global__ void computeShoalMoveKernel(float* positions, FishShoalVelocities* velocities, FishProperties* properties, int* fishIds, int* regionIndexes, int* regionStarts, int* regionsCheatSheet)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= Config::SHOAL_SIZE) return;

        int xIndex = i;
        int yIndex = i + Config::FISH_COUNT;
        int zIndex = i + Config::FISH_COUNT * 2;

        float Px = positions[xIndex];
        float Py = positions[yIndex];
        float Pz = positions[zIndex];

        float Vx = velocities->velocityX[i];
        float Vy = velocities->velocityY[i];
        float Vz = velocities->velocityZ[i];

        int numOfNeighbors = 0;

        float alignmentX = 0.0f;
        float alignmentY = 0.0f;
        float alignmentZ = 0.0f;

        float cohesionX = 0.0f;
        float cohesionY = 0.0f;
        float cohesionZ = 0.0f;

        float separationX = 0.0f;
        float separationY = 0.0f;
        float separationZ = 0.0f;
        float weightSum = 0.0f;

        int regionIndex = regionIndexes[i + Config::SHOAL_SIZE];  // use the copy of regionIndexes that is not sorted to get the region of the current fish

        // loop through neighboring regions
        int regionIterator = 0;
        int regionToCheck = regionsCheatSheet[regionIndex * 27 + regionIterator];
        while (regionToCheck != -1)
        {
            // loop through fish in the region
            int searchIndex = regionStarts[regionToCheck];
            while (searchIndex < Config::SHOAL_SIZE && regionIndexes[searchIndex] == regionToCheck) {
                int fishIndex = fishIds[searchIndex];
                if (fishIndex == i) {
                    searchIndex++;
                    continue;
                }

                float Qx = positions[fishIndex];
                float Qy = positions[fishIndex + Config::FISH_COUNT];
                float Qz = positions[fishIndex + Config::FISH_COUNT * 2];

                float distX = Qx - Px;
                float distY = Qy - Py;
                float distZ = Qz - Pz;

                float dist = sqrt(distX * distX + distY * distY + distZ * distZ);
                if (dist < 0.000001f) dist = 0.000001f;
                if (dist > properties->viewDistance) {
                    searchIndex++;
                    continue;
                }

                float denom = sqrt(Vx * Vx + Vy * Vy + Vz * Vz) * dist;
                if (denom < 0.000001f) denom = 0.000001f;
                float cosAngle = (Vx * distX + Vy * distY + Vz * distZ) / denom;
                if (cosAngle < properties->fieldOfViewCos) {
                    searchIndex++;
                    continue;
                }

                alignmentX += velocities->velocityX[fishIndex];
                alignmentY += velocities->velocityY[fishIndex];
                alignmentZ += velocities->velocityZ[fishIndex];

                cohesionX += Qx;
                cohesionY += Qy;
                cohesionZ += Qz;

                separationX += -distX / dist;
                separationY += -distY / dist;
                separationZ += -distZ / dist;
                weightSum += 1.0f / dist;

                numOfNeighbors++;
                searchIndex++;
            }

            regionIterator++;
            if (regionIterator == 27) break;
            regionToCheck = regionsCheatSheet[regionIndex * 27 + regionIterator];
		}

        if (numOfNeighbors > 0) {
            // Calculate alignment
            alignmentX = (alignmentX / numOfNeighbors - Vx);
            alignmentY = (alignmentY / numOfNeighbors - Vy);
            alignmentZ = (alignmentZ / numOfNeighbors - Vz);

            float alignment = sqrt(alignmentX * alignmentX + alignmentY * alignmentY + alignmentZ * alignmentZ);
            if (alignment > 0.0f) {
                float kA = properties->alignmentWeight * Config::ALIGNMENT_SCALE;
				alignmentX = alignmentX / alignment * kA;
				alignmentY = alignmentY / alignment * kA;
				alignmentZ = alignmentZ / alignment * kA;
			}

            // Calculate cohesion
            cohesionX = ((cohesionX / numOfNeighbors) - Px);
            cohesionY = ((cohesionY / numOfNeighbors) - Py);
            cohesionZ = ((cohesionZ / numOfNeighbors) - Pz);

            float cohesion = sqrt(cohesionX * cohesionX + cohesionY * cohesionY + cohesionZ * cohesionZ);
            if (cohesion > 0.0f) {
                float kC = properties->cohesionWeight * Config::COHESION_SCALE;
				cohesionX = cohesionX / cohesion * kC;
				cohesionY = cohesionY / cohesion * kC;
				cohesionZ = cohesionZ / cohesion * kC;
			}

            // Calculate separation
            separationX = separationX / weightSum;
            separationY = separationY / weightSum;
            separationZ = separationZ / weightSum;

            float separation = sqrt(separationX * separationX + separationY * separationY + separationZ * separationZ);
            if (separation > 0.0f) {
                float kS = properties->separationWeight * Config::SEPARATION_SCALE;
                separationX = separationX / separation * kS;
                separationY = separationY / separation * kS;
                separationZ = separationZ / separation * kS;
            }
		}

        // Calculate containment
        float containmentX = 0.0f;
        float containmentY = 0.0f;
        float containmentZ = 0.0f;

        float kF = properties->containmentWeight * Config::CONTAINMENT_SCALE;

        float dist1X = Config::AQUARIUM_SIZE / 2 - Px;
        float dist2X = Config::AQUARIUM_SIZE / 2 + Px;
        containmentX -= kF / (dist1X * dist1X);
        containmentX += kF / (dist2X * dist2X);

        float dist1Y = Config::AQUARIUM_SIZE / 2 - Py;
        float dist2Y = Config::AQUARIUM_SIZE / 2 + Py;
        containmentY -= kF / (dist1Y * dist1Y);
        containmentY += kF / (dist2Y * dist2Y);

        float dist1Z = Config::AQUARIUM_SIZE / 2 - Pz;
        float dist2Z = Config::AQUARIUM_SIZE / 2 + Pz;
        containmentZ -= kF / (dist1Z * dist1Z);
        containmentZ += kF / (dist2Z * dist2Z);

        // Calculate predator avoidance
        float predatorAvoidanceX = 0.0f;
        float predatorAvoidanceY = 0.0f;
        float predatorAvoidanceZ = 0.0f;
        weightSum = 0.0f;

        for (int predatorIndex = Config::SHOAL_SIZE; predatorIndex < Config::FISH_COUNT; predatorIndex++) {
            float Qx = positions[predatorIndex];
			float Qy = positions[predatorIndex + Config::FISH_COUNT];
			float Qz = positions[predatorIndex + Config::FISH_COUNT * 2];

			float distX = Qx - Px;
			float distY = Qy - Py;
			float distZ = Qz - Pz;

			float dist = sqrt(distX * distX + distY * distY + distZ * distZ);
			if (dist < 0.000001f) dist = 0.000001f;
			if (dist > properties->predatorViewDistance) continue;

			predatorAvoidanceX += -distX / dist;
			predatorAvoidanceY += -distY / dist;
			predatorAvoidanceZ += -distZ / dist;
            weightSum += 1.0f / dist;
        }

        float predatorAvoidance = sqrt(predatorAvoidanceX * predatorAvoidanceX + predatorAvoidanceY * predatorAvoidanceY + predatorAvoidanceZ * predatorAvoidanceZ);
        if (predatorAvoidance > 0.0f) {
			predatorAvoidanceX = predatorAvoidanceX / weightSum / predatorAvoidance * properties->predatorAvoidanceWeight;
			predatorAvoidanceY = predatorAvoidanceY / weightSum / predatorAvoidance * properties->predatorAvoidanceWeight;
			predatorAvoidanceZ = predatorAvoidanceZ / weightSum / predatorAvoidance * properties->predatorAvoidanceWeight;
		}

        // Calculate net force
        float FSx = alignmentX + cohesionX + separationX + containmentX + predatorAvoidanceX;
        float FSy = alignmentY + cohesionY + separationY + containmentY + predatorAvoidanceY;
        float FSz = alignmentZ + cohesionZ + separationZ + containmentZ + predatorAvoidanceZ;

        float force = sqrt(FSx * FSx + FSy * FSy + FSz * FSz);
        if (force != 0.0f) {
            float clampedFroce = force > properties->maxForce ? properties->maxForce : force;

            float Fx = FSx * clampedFroce / force;
            float Fy = FSy * clampedFroce / force;
            float Fz = FSz * clampedFroce / force;

            float aX = Fx / properties->mass;
            float aY = Fy / properties->mass;
            float aZ = Fz / properties->mass;

            Vx += aX;
            Vy += aY;
            Vz += aZ;
		}

        // Limit speed
        float speed = sqrt(Vx * Vx + Vy * Vy + Vz * Vz);
        if (speed > properties->maxSpeed) {
            Vx = Vx * properties->maxSpeed / speed;
            Vy = Vy * properties->maxSpeed / speed;
            Vz = Vz * properties->maxSpeed / speed;
        }

        float newPx = Px + Vx;
        if (Config::AQUARIUM_SIZE / 2 - newPx < 0.001 || Config::AQUARIUM_SIZE / 2 - newPx > 1.999) 
        {
			Vx = 0;
		}

        float newPy = Py + Vy;
        if (Config::AQUARIUM_SIZE / 2 - newPy < 0.001 || Config::AQUARIUM_SIZE / 2 - newPy > 1.999) 
		{
            Vy = 0;
        }

        float newPz = Pz + Vz;
        if (Config::AQUARIUM_SIZE / 2 - newPz < 0.001 || Config::AQUARIUM_SIZE / 2 - newPz > 1.999)
        {
			Vz = 0;
		}

        positions[xIndex] += Vx;
        positions[yIndex] += Vy;
        positions[zIndex] += Vz;

        velocities->velocityX[i] = Vx;
        velocities->velocityY[i] = Vy;
        velocities->velocityZ[i] = Vz;
    }

    __global__ void computePredatorMoveKernel(float* positions, PredatorVelocities* velocities) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= Config::PREDATOR_COUNT) return;

		int xIndex = i + Config::SHOAL_SIZE;
		int yIndex = xIndex + Config::FISH_COUNT;
		int zIndex = yIndex + Config::FISH_COUNT;

		float Px = positions[xIndex];
		float Py = positions[yIndex];
		float Pz = positions[zIndex];

		float Vx = velocities->velocityX[i];
        float Vy = velocities->velocityY[i];
        float Vz = velocities->velocityZ[i];

        float newPx = Px + Vx;
        if (Config::AQUARIUM_SIZE / 2 - newPx < 0.001 || Config::AQUARIUM_SIZE / 2 - newPx > 1.999)
        {
            Vx = -Vx;
        }

        float newPy = Py + Vy;
        if (Config::AQUARIUM_SIZE / 2 - newPy < 0.001 || Config::AQUARIUM_SIZE / 2 - newPy > 1.999)
        {
            Vy = -Vy;
        }

        float newPz = Pz + Vz;
        if (Config::AQUARIUM_SIZE / 2 - newPz < 0.001 || Config::AQUARIUM_SIZE / 2 - newPz > 1.999)
        {
            Vz = -Vz;
        }

        positions[xIndex] += Vx;
        positions[yIndex] += Vy;
        positions[zIndex] += Vz;

        velocities->velocityX[i] = Vx;
        velocities->velocityY[i] = Vy;
        velocities->velocityZ[i] = Vz;
    }
}
